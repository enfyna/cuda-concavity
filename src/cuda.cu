#include "hip/hip_runtime.h"
#include <cassert>
#include <cstddef>
#include <cstdio>
#include <cstdlib>
#include <ctime>

#include "raylib.h"

__global__ void calculate_angles(int* dest, size_t count, Vector2* points)
{
    Vector2 l, p, r;
    size_t c = blockIdx.x;

    l = points[(c + count - 1) % count];
    p = points[c % count];
    r = points[(c + 1) % count];
    Vector2 ps[] = { l, p, r };

    // Vector2Subtract(l, p);
    Vector2 ba = (Vector2) { .x = l.x - p.x, .y = l.y - p.y };
    // Vector2Subtract(r, p);
    Vector2 bc = (Vector2) { .x = r.x - p.x, .y = r.y - p.y };

    float cross = ba.x * bc.y - ba.y * bc.x;

    dest[blockIdx.x] = (cross >= 0) ? 1 : -1;
    printf("block: %u, thread: %u => %d\n"
           "%5.1f,%5.1f|"
           "%5.1f,%5.1f|"
           "%5.1f,%5.1f|\n",
        blockIdx.x, threadIdx.x, dest[blockIdx.x], ps[0].x, ps[0].y, ps[1].x, ps[1].y, ps[2].x, ps[2].y);
}

extern "C" {

bool is_concave(size_t count, Vector2* points)
{
    hipError_t err;

    printf("%zu:\n", count);
    for (size_t i = 0; i < count; i++) {
        printf("%f, %f\n", points[i].x, points[i].y);
    }

    int size = count * sizeof(Vector2);
    int size_dest = count * sizeof(int);

    Vector2* d_a;
    err = hipMalloc(&d_a, size);
    if (err != hipSuccess) {
        printf("hipMalloc failed: %s\n", hipGetErrorString(err));
        fprintf(stderr, "err: %d\n", err);
        exit(1);
    }

    int* d_dest;
    err = hipMalloc(&d_dest, size_dest);
    if (err != hipSuccess) {
        printf("hipMalloc failed: %s\n", hipGetErrorString(err));
        fprintf(stderr, "err: %d\n", err);
        exit(1);
    }

    err = hipMemcpy(d_a, points, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("hipMemcpy failed: %s\n", hipGetErrorString(err));
        fprintf(stderr, "err: %d\n", err);
        exit(1);
    }

    calculate_angles<<<count, 1>>>(d_dest, count, d_a);
    hipDeviceSynchronize();

    int dest[count];
    hipMemcpy(dest, d_dest, size_dest, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_dest);

    for (size_t i = 1; i <= count; i++) {
        if (dest[i % count] + dest[i - 1] == 0) {
            printf("\n");
            return true;
        }
        printf("%d,", dest[i % count]);
    }
    printf("\n");

    return false;
}
}
